#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <cuda_pipeline_primitives.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <random>
#include <stdexcept>
#include <vector>

#include "util/error.h"
#include "util/util.cuh"

const int limit = 32;

__global__ void fp32_naive_mm(const float* A, const float* B, float* C, int M, int N, int K)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  int m = blockIdx.y * blockDim.y + threadIdx.y;

  if (n >= N || m >= M) {
    return;
  }

  A += m * K;
  B += n;
  float sum = 0.0;
#pragma unroll
  for (int k = 0; k < K; ++k) {
    sum += A[k] * B[k * N];
  }
  C[m * N + n] = sum;
}

void launch_fp32_naive_mm(const float* A, const float* B, float* C, int M, int N, int K)
{
  dim3 threads_per_block(16, 16);
  dim3 blocks_per_grid((N + threads_per_block.x - 1) / threads_per_block.x,
                       (M + threads_per_block.y - 1) / threads_per_block.y);

  fp32_naive_mm<<<blocks_per_grid, threads_per_block>>>(A, B, C, M, N, K);
  CHECK_CUDA_ERROR();
}

template<typename T, int BLOCK_TILE_M, int BLOCK_TILE_N, int WARP_TILE_M, int WARP_TILE_N, int TILE_K>
__global__ void llmmm_fp16_mma_m8n8k4(const T* A, const T* B, T* C, int M, int N, int K)
{
  static_assert(std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value);
  const int     warp_id                   = threadIdx.x / 32;
  const int     lane_id                   = threadIdx.x % 32;
  const int     mma_m8n8k4_computation_id = lane_id % 16 / 4;
  const int     mma_m8n8k4_lane_id        = lane_id % 4 + lane_id / 16 * 4;
  const int     mma_m8n8k4_group_id       = lane_id / 16;  // low or high
  constexpr int M_COMPUTATION_COUNT       = WARP_TILE_M / 8;
  constexpr int N_COMPUTATION_COUNT       = WARP_TILE_N / 8;
  static_assert(M_COMPUTATION_COUNT * N_COMPUTATION_COUNT == 4);

  const int m_block_offset = blockIdx.y * BLOCK_TILE_M;
  const int n_block_offset = blockIdx.x * BLOCK_TILE_N;
  const int m_warp_offset  = warp_id * WARP_TILE_M;

  static_assert(TILE_K % 8 == 0);
  static_assert(BLOCK_TILE_N % 8 == 0);
  __shared__ T A_sm[TILE_K / 8][BLOCK_TILE_M][8];
  __shared__ T B_sm[BLOCK_TILE_N / 8][TILE_K][8];

  constexpr int WARP_COUNT   = BLOCK_TILE_M / WARP_TILE_M;
  constexpr int THREAD_COUNT = WARP_COUNT * 32;

  static_assert(TILE_K * BLOCK_TILE_M / 2 % THREAD_COUNT == 0);
  static_assert(TILE_K * BLOCK_TILE_N / 2 % THREAD_COUNT == 0);

  constexpr int A_LDG_REG_COUNT = TILE_K * BLOCK_TILE_M / THREAD_COUNT;
  constexpr int B_LDG_REG_COUNT = TILE_K * BLOCK_TILE_N / THREAD_COUNT;
  T             A_ldg_reg[A_LDG_REG_COUNT];
  T             B_ldg_reg[B_LDG_REG_COUNT];
  static_assert(A_LDG_REG_COUNT % 8 == 0);
  static_assert(B_LDG_REG_COUNT % 8 == 0);
  constexpr int A_LDG_LOOP_COUNT = A_LDG_REG_COUNT / 8;
  constexpr int B_LDG_LOOP_COUNT = B_LDG_REG_COUNT / 8;

  T     A_mma_reg[2][4];
  T     B_mma_reg[4];
  float C_mma_reg[BLOCK_TILE_N / WARP_TILE_N][8] = {0};

  for (int k_offset = 0; k_offset < K; k_offset += TILE_K) {
    for (int loop = 0; loop < A_LDG_LOOP_COUNT; ++loop) {
      const int index = (loop * THREAD_COUNT + threadIdx.x) * 8;
      const int m     = index / TILE_K;
      const int k     = index % TILE_K;
      FETCH_FLOAT4(A_ldg_reg[loop * 8], A[OFFSET(m_block_offset + m, k_offset + k, K)]);
      STORE_FLOAT4(A_sm[k / 8][m][0], A_ldg_reg[loop * 8]);
    }

    for (int loop = 0; loop < B_LDG_LOOP_COUNT; ++loop) {
      const int index = (loop * THREAD_COUNT + threadIdx.x) * 8;
      const int n     = index % BLOCK_TILE_N;
      const int k     = index / BLOCK_TILE_N;
      FETCH_FLOAT4(B_ldg_reg[loop * 8], B[OFFSET(k_offset + k, n_block_offset + n, N)]);
      STORE_FLOAT4(B_sm[n / 8][k][0], B_ldg_reg[loop * 8]);
    }

    __syncthreads();

#pragma unroll
    for (int mma_k_offset = 0; mma_k_offset < TILE_K; mma_k_offset += 8) {
      FETCH_FLOAT4(A_mma_reg[0],
                   A_sm[mma_k_offset / 8]
                       [m_warp_offset + mma_m8n8k4_computation_id / M_COMPUTATION_COUNT * 8 + mma_m8n8k4_lane_id][0]);
#pragma unroll
      for (int mma_n_offset = 0; mma_n_offset < BLOCK_TILE_N; mma_n_offset += WARP_TILE_N) {
        FETCH_FLOAT(B_mma_reg[0],
                    B_sm[mma_n_offset / 8 + mma_m8n8k4_computation_id % M_COMPUTATION_COUNT]
                        [mma_k_offset + mma_m8n8k4_lane_id % 4][mma_m8n8k4_group_id * 2]);
        FETCH_FLOAT(B_mma_reg[2],
                    B_sm[mma_n_offset / 8 + mma_m8n8k4_computation_id % M_COMPUTATION_COUNT]
                        [mma_k_offset + mma_m8n8k4_lane_id % 4][mma_m8n8k4_group_id * 2 + 4]);
        mma_sync_aligned_m8n8k4_row_row_f32_f16_f16_f32<T>(
          C_mma_reg[mma_n_offset / WARP_TILE_N], A_mma_reg[0], B_mma_reg, C_mma_reg[mma_n_offset / WARP_TILE_N]);

        FETCH_FLOAT(B_mma_reg[0],
                    B_sm[mma_n_offset / 8 + mma_m8n8k4_computation_id % M_COMPUTATION_COUNT]
                        [mma_k_offset + mma_m8n8k4_lane_id % 4 + 4][mma_m8n8k4_group_id * 2]);
        FETCH_FLOAT(B_mma_reg[2],
                    B_sm[mma_n_offset / 8 + mma_m8n8k4_computation_id % M_COMPUTATION_COUNT]
                        [mma_k_offset + mma_m8n8k4_lane_id % 4 + 4][mma_m8n8k4_group_id * 2 + 4]);
        mma_sync_aligned_m8n8k4_row_row_f32_f16_f16_f32<T>(
          C_mma_reg[mma_n_offset / WARP_TILE_N], A_mma_reg[1], B_mma_reg, C_mma_reg[mma_n_offset / WARP_TILE_N]);
      }
    }
    __syncthreads();
  }
  const int m_computation_offset = mma_m8n8k4_computation_id / M_COMPUTATION_COUNT * 8;
  const int n_computation_stride = mma_m8n8k4_computation_id % N_COMPUTATION_COUNT * 8;
  int       m_global             = m_block_offset + m_warp_offset + m_computation_offset + (mma_m8n8k4_lane_id & 0xfd);
  T         C_reg[8];
  for (int n_offset = 0; n_offset < BLOCK_TILE_N; n_offset += WARP_TILE_N) {
    for (int i = 0; i < 8; ++i) {
      C_reg[i] = C_mma_reg[n_offset / WARP_TILE_N][i];
    }
    int n_global = n_block_offset + n_offset + (mma_m8n8k4_lane_id & 0x2) * 2 + n_computation_stride;
    STORE_FLOAT2(C[OFFSET(m_global, n_global, N)], C_reg[0]);
    STORE_FLOAT2(C[OFFSET(m_global + 2, n_global, N)], C_reg[4]);
  }
}

template<typename T>
__global__ void llmmm_fp16_mma_m16n8k8(const T* A, const T* B, T* C, int M, int N, int K)
{
}

template<typename T>
__global__ void llmmm_fp16_mma_m16n8k16(const T* A, const T* B, T* C, int M, int N, int K)
{
}

template<typename T>
void launch_llmmm_fp16_mma_m8n8k4(const T* A, const T* B, T* C, int M, int N, int K)
{
  constexpr int BLOCK_TILE_M = 128;
  constexpr int BLOCK_TILE_N = 128;
  constexpr int TILE_K       = 16;
  if (!(M % BLOCK_TILE_M == 0 && N % BLOCK_TILE_N == 0 && K % TILE_K == 0)) {
    throw std::runtime_error("M or N or K are not aligned.");
  }
  static_assert(16 <= BLOCK_TILE_M && BLOCK_TILE_M <= 128 && (BLOCK_TILE_M & (BLOCK_TILE_M - 1)) == 0);
  static_assert(16 <= BLOCK_TILE_N && BLOCK_TILE_N <= 256 && (BLOCK_TILE_N & (BLOCK_TILE_N - 1)) == 0);
  static_assert(TILE_K % 4 == 0 && 16 <= TILE_K && TILE_K <= 128 && ((TILE_K & (TILE_K - 1)) == 0));
  constexpr int m_per_warp = 16;
  constexpr int n_per_warp = 16;
  static_assert(BLOCK_TILE_M % m_per_warp == 0 && BLOCK_TILE_N % n_per_warp == 0);
  static_assert(m_per_warp == 8 || m_per_warp == 16 || m_per_warp == 32);
  static_assert(n_per_warp == 8 || n_per_warp == 16 || n_per_warp == 32);
  static_assert(m_per_warp * n_per_warp == 8 * 8 * 4);
  constexpr int warp_count = BLOCK_TILE_M / m_per_warp;
  static_assert(1 <= warp_count && warp_count <= 16 && (warp_count & (warp_count - 1)) == 0);
  dim3 grid(N / BLOCK_TILE_N, M / BLOCK_TILE_M);
  dim3 block(warp_count * 32);
  llmmm_fp16_mma_m8n8k4<T, BLOCK_TILE_M, BLOCK_TILE_N, m_per_warp, n_per_warp, TILE_K>
    <<<grid, block>>>(A, B, C, M, N, K);
  CHECK_CUDA_ERROR();
}

template<typename T, typename = std::enable_if_t<std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value>>
int test()
{
  static const int M = (1 << 12), N = (1 << 12), K = (1 << 12);
  // static const int M = 128, N = 128, K = 128;

  std::vector<float>                    host_A(M * K), host_B(K * N), host_C(M * N);
  std::vector<T>                        host_fp16_A(M * K), host_fp16_B(K * N), host_fp16_C(M * N);
  std::random_device                    rd;
  std::mt19937                          gen(rd());
  std::uniform_real_distribution<float> dis(-1, 1);
  for (auto& vec : {&host_A, &host_B}) {
#if 1
    for (auto& data : *vec) {
      data = dis(gen);
    }
#else
    if (vec == &host_A) {
      for (size_t i = 0; i < vec->size(); ++i) {
        int row            = i / K;
        int col            = i % K;
        vec->operator[](i) = (row == col);
      }
    }
    if (vec == &host_B) {
      for (size_t i = 0; i < vec->size(); ++i) {
        int row = i / N;
        int col = i % N;
        if (row < limit && col < limit) {
          vec->operator[](i) = row * limit + col;
        }
        else {
          vec->operator[](i) = 0;
        }
      }
    }
#endif
  }
  for (auto [fp32, fp16] : {std::make_pair(&host_A, &host_fp16_A),
                            std::make_pair(&host_B, &host_fp16_B),
                            std::make_pair(&host_C, &host_fp16_C)}) {
    for (int i = 0; i < fp16->size(); ++i) {
      fp16->at(i) = T(fp32->at(i));
    }
  }

  float *A, *B, *C;
  for (auto& pair : {std::make_pair(host_A, &A), std::make_pair(host_B, &B), std::make_pair(host_C, &C)}) {
    const std::vector<float>& host   = pair.first;
    float*&                   device = *pair.second;
    hipMalloc(&device, sizeof(float) * host.size());
    hipMemcpy(device, host.data(), sizeof(float) * host.size(), hipMemcpyDefault);
    CHECK_CUDA_ERROR();
  }

  {
    hipMemset(C, 0, M * N * sizeof(float));
    launch_fp32_naive_mm(A, B, C, M, N, K);
    hipMemcpy(host_C.data(), C, sizeof(float) * host_C.size(), hipMemcpyDefault);
    CHECK_CUDA_ERROR();
  }

  T *fp16_A, *fp16_B, *fp16_C;
  for (auto& pair : {std::make_pair(host_fp16_A, &fp16_A),
                     std::make_pair(host_fp16_B, &fp16_B),
                     std::make_pair(host_fp16_C, &fp16_C)}) {
    const std::vector<T>& host   = pair.first;
    T*&                   device = *pair.second;
    hipMalloc(&device, sizeof(T) * host.size());
    hipMemcpy(device, host.data(), sizeof(T) * host.size(), hipMemcpyDefault);
    CHECK_CUDA_ERROR();
  }

  launch_llmmm_fp16_mma_m8n8k4(fp16_A, fp16_B, fp16_C, M, N, K);
  hipMemcpy(host_fp16_C.data(), fp16_C, sizeof(T) * host_fp16_C.size(), hipMemcpyDefault);
  for (int i = 0; i < 16; ++i) {
    for (int j = 0; j < 16; ++j) {
      printf("%8.3f ", float(host_fp16_C[i * N + j]));
    }
    printf("\n");
  }
  for (int i = 0; i < host_fp16_C.size(); ++i) {
    if (fabs(float(host_fp16_C[i]) - host_C[i]) > 1e-1) {
      printf("i = %d, fp16 = %8.3f, fp32 = %8.3f\n", i, float(host_fp16_C[i]), host_C[i]);
      break;
    }
  }

  return 0;
}

int main()
{
  test<half>();
  // test<__hip_bfloat16>();
  return 0;
}
