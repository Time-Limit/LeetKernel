#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <cuda_pipeline_primitives.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <random>
#include <stdexcept>
#include <type_traits>
#include <vector>

#include "util/error.h"
#include "util/util.cuh"

const int limit = 16;

__global__ void fp32_naive_mm(const float* A, const float* B, float* C, int M, int N, int K)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  int m = blockIdx.y * blockDim.y + threadIdx.y;

  if (n >= N || m >= M) {
    return;
  }

  A += m * K;
  B += n;
  double sum = 0.0;
#pragma unroll
  for (int k = 0; k < K; ++k) {
    sum += A[k] * B[k * N];
  }
  C[m * N + n] = sum;
}

void launch_fp32_naive_mm(const float* A, const float* B, float* C, int M, int N, int K)
{
  dim3 threads_per_block(16, 16);
  dim3 blocks_per_grid((N + threads_per_block.x - 1) / threads_per_block.x,
                       (M + threads_per_block.y - 1) / threads_per_block.y);

  fp32_naive_mm<<<blocks_per_grid, threads_per_block>>>(A, B, C, M, N, K);
  CHECK_CUDA_ERROR();
}

template<typename T, int BLOCK_TILE_M, int BLOCK_TILE_N, int WARP_TILE_M, int WARP_TILE_N>
__global__ void fp16_mma_m8n8k16_ldmatrix(const T* A, const T* B, const T* C, int M, int N, int K)
{
  constexpr int WARP_COUNT   = BLOCK_TILE_M / WARP_TILE_M * BLOCK_TILE_N / WARP_TILE_N;
  constexpr int THREAD_COUNT = WARP_COUNT * 32;

  constexpr int LOOP_TILE_K = 16;
  // The 64 elements of type T in each 8x8 matrix are stored consecutively in a single layer.
  __shared__ T A_sm[2][BLOCK_TILE_M][LOOP_TILE_K / 2];
  __shared__ T B_sm[2][BLOCK_TILE_N][LOOP_TILE_K / 2];

  static_assert(BLOCK_TILE_M * LOOP_TILE_K % THREAD_COUNT == 0);
  static_assert(BLOCK_TILE_M * LOOP_TILE_K / THREAD_COUNT % 4 == 0);
  constexpr int A_LDG_COUNT_PER_THREAD = BLOCK_TILE_M * LOOP_TILE_K / THREAD_COUNT;
  constexpr int A_LDG_LOOP_COUNT       = A_LDG_COUNT_PER_THREAD / 4;
  // clang-format off
  // This is the thread layout of the same warp that loads matrix A, where each thread reads M1xK4 elements of type T at a
  // loop iteration.
  // T0 T8  .... T24
  // T1 T9       T25
  // T2 T10      T26
  // T3 T11      T27
  // T4 T12      T28
  // T5 T13      T29
  // T6 T14      T30
  // T7 T15      T31
  // clang-format on
  T A_ldg_reg[A_LDG_LOOP_COUNT][4];

  static_assert(BLOCK_TILE_N * LOOP_TILE_K % THREAD_COUNT == 0);
  static_assert(BLOCK_TILE_N * LOOP_TILE_K / THREAD_COUNT % 4 == 0);
  constexpr int B_LDG_COUNT_PER_THREAD = BLOCK_TILE_N * LOOP_TILE_K / THREAD_COUNT;
  constexpr int B_LDG_LOOP_COUNT       = B_LDG_COUNT_PER_THREAD / 4;
  // clang-format off
  // This is the thread layout of the same warp that loads matrix B, where each thread reads K2xN2 elements of type T at a
  // loop iteration.
  // T0 T4 ... T28
  // T1 T5     T29
  // T2 T6     T30
  // T3 T7     T31
  // clang-format on
  T B_ldg_reg[B_LDG_LOOP_COUNT][2][2];

  const int m_block_offset = blockIdx.y * BLOCK_TILE_M;
  const int n_block_offset = blockIdx.x * BLOCK_TILE_N;

  const int warp_id = threadIdx.x / 32;
  const int lane_id = threadIdx.x % 32;

  constexpr int M_MMA_WARP_COUNT       = BLOCK_TILE_M / WARP_TILE_M;
  constexpr int M_GROUP_COUNT_PER_WARP = WARP_TILE_M / 8;
  constexpr int N_GROUP_COUNT_PER_WARP = WARP_TILE_N / 16;

  T     A_mma_reg[M_GROUP_COUNT_PER_WARP][4];
  T     B_mma_reg[N_GROUP_COUNT_PER_WARP][8];
  float C_mma_reg[M_GROUP_COUNT_PER_WARP][N_GROUP_COUNT_PER_WARP][4] = {0};

  const int m_warp_offset = warp_id % M_MMA_WARP_COUNT * WARP_TILE_M;
  const int n_warp_offset = warp_id / M_MMA_WARP_COUNT * WARP_TILE_N;

  for (int k_loop_offset = 0; k_loop_offset < K; k_loop_offset += LOOP_TILE_K) {
    for (int loop = 0; loop < A_LDG_LOOP_COUNT; ++loop) {
      const int m = (loop * WARP_COUNT + warp_id) * 8 + lane_id % 8;
      const int k = lane_id / 8 * 4;
      FETCH_FLOAT2(A_ldg_reg[loop][0], A[OFFSET(m_block_offset + m, k_loop_offset + k, K)]);
    }
    for (int loop = 0; loop < B_LDG_LOOP_COUNT; ++loop) {
      const int k = (loop * WARP_COUNT + warp_id) % 2 * 8 + lane_id % 4 * 2;
      const int n = (loop * WARP_COUNT + warp_id) / 2 * 16 + lane_id / 4 * 2;
      FETCH_FLOAT(B_ldg_reg[loop][0][0], B[OFFSET(k_loop_offset + k, n_block_offset + n, K)]);
      FETCH_FLOAT(B_ldg_reg[loop][1][0], B[OFFSET(k_loop_offset + k + 1, n_block_offset + n, K)]);
    }
    for (int loop = 0; loop < A_LDG_LOOP_COUNT; ++loop) {
      const int m = (loop * WARP_COUNT + warp_id) * 8 + lane_id % 8;
      const int k = lane_id / 8 * 4;
      STORE_FLOAT2(A_sm[k / 8][m][k % 8], A_ldg_reg[loop][0]);
    }
    for (int loop = 0; loop < B_LDG_LOOP_COUNT; ++loop) {
      const int k = (loop * WARP_COUNT + warp_id) % 2 * 8 + lane_id % 4 * 2;
      const int n = (loop * WARP_COUNT + warp_id) / 2 * 16 + lane_id / 4 * 2;
      {
        T transpose[2] = {B_ldg_reg[loop][0][0], B_ldg_reg[loop][1][0]};
        STORE_FLOAT(B_sm[k / 8][n][k % 8], transpose[0]);
      }
      {
        T transpose[2] = {B_ldg_reg[loop][0][1], B_ldg_reg[loop][1][1]};
        STORE_FLOAT(B_sm[k / 8][n + 1][k % 8], transpose[0]);
      }
    }
    __syncthreads();
    // if (k_loop_offset == -1 && this_thread_can_log(0)) {
    //   const T* data = &A_sm[0][0][0];
    //   for (int i = 0; i < BLOCK_TILE_M * LOOP_TILE_K; ++i) {
    //     if (i % 64 == 0) {
    //       printf("\nA_sm, layer = %03d, ", i / 64);
    //     }
    //     printf("m%03dk%03d ", int(data[i]) / limit, int(data[i]) % limit);
    //   }
    //   data = &B_sm[0][0][0];
    //   for (int i = 0; i < BLOCK_TILE_N * LOOP_TILE_K; ++i) {
    //     if (i % 64 == 0) {
    //       printf("\nB_sm, layer = %03d, ", i / 64);
    //     }
    //     printf("n%03dk%03d ", int(data[i]) % limit, int(data[i]) / limit);
    //   }
    // }
    // The shape of A group is m8xk16
    for (int group = 0; group < M_GROUP_COUNT_PER_WARP; ++group) {
      uint32_t src = __cvta_generic_to_shared(&A_sm[lane_id / 8][m_warp_offset + group * 8 + lane_id % 8][0]);
      asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];"
                   : "=r"(*(uint32_t*)&A_mma_reg[group][0]), "=r"(*(uint32_t*)&A_mma_reg[group][2])
                   : "r"(src));
    }
    // The shape of B group is n16xk16
    for (int group = 0; group < N_GROUP_COUNT_PER_WARP; ++group) {
      uint32_t src = __cvta_generic_to_shared(&B_sm[lane_id / 16][n_warp_offset + group * 16 + lane_id % 16][0]);
      asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];"
                   : "=r"(*(uint32_t*)&B_mma_reg[group][0]),
                     "=r"(*(uint32_t*)&B_mma_reg[group][2]),
                     "=r"(*(uint32_t*)&B_mma_reg[group][4]),
                     "=r"(*(uint32_t*)&B_mma_reg[group][6])
                   : "r"(src));
    }
    // for (int i = 0; k_loop_offset == -1 && i < 256; ++i) {
    //   if (this_thread_can_log(i)) {
    //     for (int group = 0; group < M_GROUP_COUNT_PER_WARP; ++group) {
    //       printf("\nA_mma_reg, thread = %03d, group = %03d, ", threadIdx.x, group);
    //       for (int r = 0; r < 4; ++r) {
    //         printf(" m%03dk%03d", int(A_mma_reg[group][r]) / limit, int(A_mma_reg[group][r]) % limit);
    //       }
    //     }
    //   }
    //   __syncthreads();
    // }
    // for (int i = 0; k_loop_offset == -1 && i < 256; ++i) {
    //   if (this_thread_can_log(i)) {
    //     for (int group = 0; group < N_GROUP_COUNT_PER_WARP; ++group) {
    //       printf("\nB_mma_reg, thread = %03d, group = %03d, ", threadIdx.x, group);
    //       for (int r = 0; r < 8; ++r) {
    //         printf(" n%03dk%03d", int(B_mma_reg[group][r]) % limit, int(B_mma_reg[group][r]) / limit);
    //       }
    //     }
    //   }
    //   __syncthreads();
    // }
    for (int mg = 0; mg < M_GROUP_COUNT_PER_WARP; ++mg) {
      for (int ng = 0; ng < N_GROUP_COUNT_PER_WARP; ++ng) {
        mma_m16n8k16_row_col(C_mma_reg[mg][ng], B_mma_reg[ng], A_mma_reg[mg], C_mma_reg[mg][ng]);
      }
    }
    __syncthreads();
  }

  for (int mg = 0; mg < M_GROUP_COUNT_PER_WARP; ++mg) {
    for (int ng = 0; ng < N_GROUP_COUNT_PER_WARP; ++ng) {
      // int m = m_block_offset + m_warp_offset + mg * 8;
      // int n = n_block_offset + n_warp_offset + ng * 16;
      T casted[4] = {
        C_mma_reg[mg][ng][0],
        C_mma_reg[mg][ng][1],
        C_mma_reg[mg][ng][2],
        C_mma_reg[mg][ng][3],
      };
      uint32_t& first  = *(uint32_t*)(&casted[0]);
      uint32_t& second = *(uint32_t*)(&casted[2]);
      uint32_t  swap   = (first ^ second) * (!(lane_id & 0x4));
      first ^= swap;
      second ^= swap;
      first = __shfl_xor_sync(0xffffffff, first, 0x4);
      swap  = (first ^ second) * (!(lane_id & 0x4));
      first ^= swap;
      second ^= swap;
      T                store[4]           = {casted[0], casted[2], casted[1], casted[3]};
      static const int lane_2_n_offset[8] = {0, 8, 2, 10, 4, 12, 6, 14};
      // int              m                  = m_block_offset + m_loop * LOOP_TILE_M + lane_id % 4 * 2;
      // int              n                  = n_block_offset + n_warp_offset + lane_2_n_offset[lane_id / 4];
      int m = m_block_offset + m_warp_offset + mg * 8 + lane_id % 4 * 2;
      int n = n_block_offset + n_warp_offset + ng * 16 + lane_2_n_offset[lane_id / 4];
      STORE_FLOAT(C[OFFSET(m, n, N)], store[0]);
      STORE_FLOAT(C[OFFSET(m + 1, n, N)], store[2]);
    }
  }
}

#define define_check_function(function)                                                                                                                  \
  template<typename T, int BLOCK_TILE_M, int BLOCK_TILE_N, int WARP_TILE_M, int WARP_TILE_N>                                                             \
  void launch_##function(const T* A, const T* B, T* C, int M, int N, int K)                                                                              \
  {                                                                                                                                                      \
    if (std::is_same<T, half>::value == false && std::is_same<T, __hip_bfloat16>::value == false) {                                                       \
      throw std::runtime_error("T is not supported.");                                                                                                   \
    }                                                                                                                                                    \
    constexpr int LOOP_TILE_K = 16;                                                                                                                      \
    if (!(M % BLOCK_TILE_M == 0 && N % BLOCK_TILE_N == 0 && K % LOOP_TILE_K == 0)) {                                                                     \
      throw std::runtime_error("M or N or K are not aligned.");                                                                                          \
    }                                                                                                                                                    \
    static_assert(8 <= BLOCK_TILE_M && BLOCK_TILE_M <= 256 && (BLOCK_TILE_M & (BLOCK_TILE_M - 1)) == 0);                                                 \
    static_assert(16 <= BLOCK_TILE_N && BLOCK_TILE_N <= 256 && (BLOCK_TILE_N & (BLOCK_TILE_N - 1)) == 0);                                                \
    static_assert(LOOP_TILE_K == 16);                                                                                                                    \
    static_assert(BLOCK_TILE_M % WARP_TILE_M == 0 && BLOCK_TILE_N % WARP_TILE_N == 0);                                                                   \
    static_assert(WARP_TILE_N % 16 == 0 && WARP_TILE_M % 8 == 0 && WARP_TILE_N / 16 == WARP_TILE_M / 8);                                                 \
    constexpr int WARP_COUNT = BLOCK_TILE_N / WARP_TILE_N * BLOCK_TILE_M / WARP_TILE_M;                                                                  \
    static_assert(1 <= WARP_COUNT && WARP_COUNT <= 32 && (WARP_COUNT & (WARP_COUNT - 1)) == 0);                                                          \
    dim3 grid(N / BLOCK_TILE_N, M / BLOCK_TILE_M);                                                                                                       \
    dim3 block(WARP_COUNT * 32);                                                                                                                         \
    function<T, BLOCK_TILE_M, BLOCK_TILE_N, WARP_TILE_M, WARP_TILE_N><<<grid, block>>>(A, B, C, M, N, K);                                                \
    CHECK_CUDA_ERROR();                                                                                                                                  \
  }                                                                                                                                                      \
  template<typename T>                                                                                                                                   \
  void function##___check_relative_error(                                                                                                                \
    const T* A, const T* B, T* C, int M, int N, int K, const std::vector<float>& base)                                                                   \
  {                                                                                                                                                      \
    std::vector<T> host_C(M* N);                                                                                                                         \
    memset(host_C.data(), 0, sizeof(T) * host_C.size());                                                                                                 \
    launch_##function<T, 128, 128, 32, 64>(A, B, C, M, N, K);                                                                                            \
    hipMemcpy(host_C.data(), C, sizeof(T) * host_C.size(), hipMemcpyDefault);                                                                          \
    float max_error = 0, base_value, current_value;                                                                                                      \
    int   position  = 0;                                                                                                                                 \
    for (int i = 0; i < host_C.size(); ++i) {                                                                                                            \
      if (fabs(float(host_C[i]) - base[i]) > max_error) {                                                                                                \
        max_error     = fabs(float(host_C[i]) - base[i]);                                                                                                \
        base_value    = base[i];                                                                                                                         \
        current_value = host_C[i];                                                                                                                       \
        position      = i;                                                                                                                               \
      }                                                                                                                                                  \
    }                                                                                                                                                    \
    const char* type = std::is_same<T, half>::value ? "half" : "__hip_bfloat16";                                                                          \
    const char* name = #function;                                                                                                                        \
    printf(                                                                                                                                              \
      "max_relative_error = %8.6f, max_absolute_error = %8.3f, base_value = %10.3f, current_value = %10.3f, type=%16s, function=%s\n", \
      fabs(max_error / base_value),                                                                                                                      \
      max_error,                                                                                                                                         \
      base_value,                                                                                                                                        \
      current_value,                                                                                                                                     \
      type,                                                                                                                                              \
      name);                                                                                                                                             \
  }

/* clang-format off */
define_check_function(fp16_mma_m8n8k16_ldmatrix);
/* clang-format on */

template<typename T, typename = std::enable_if_t<std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value>>
int test(const std::vector<float>& host_A,
         const std::vector<float>& host_B,
         const std::vector<float>& host_C,
         int                       M,
         int                       N,
         int                       K)
{
  std::vector<T> host_fp16_A(M * K), host_fp16_B(K * N), host_fp16_C(M * N);
  for (auto [fp32, fp16] : {std::make_pair(&host_A, &host_fp16_A),
                            std::make_pair(&host_B, &host_fp16_B),
                            std::make_pair(&host_C, &host_fp16_C)}) {
    for (int i = 0; i < fp16->size(); ++i) {
      fp16->at(i) = T(fp32->at(i));
    }
  }

  T *fp16_A, *fp16_B, *fp16_C;
  for (auto& pair : {std::make_pair(host_fp16_A, &fp16_A),
                     std::make_pair(host_fp16_B, &fp16_B),
                     std::make_pair(host_fp16_C, &fp16_C)}) {
    const std::vector<T>& host   = pair.first;
    T*&                   device = *pair.second;
    hipMalloc(&device, sizeof(T) * host.size());
    hipMemcpy(device, host.data(), sizeof(T) * host.size(), hipMemcpyDefault);
    CHECK_CUDA_ERROR();
  }

  /* clang-format off */
  fp16_mma_m8n8k16_ldmatrix___check_relative_error(fp16_A, fp16_B, fp16_C, M, N, K, host_C);
  /* clang-format on */

  CHECK_CUDA_RETURN(hipFree(fp16_A));
  CHECK_CUDA_RETURN(hipFree(fp16_B));
  CHECK_CUDA_RETURN(hipFree(fp16_C));

  return 0;
}

int main()
{
  static const int M = (1 << 12), N = (1 << 12), K = (1 << 12);
  // static const int M = 128, N = 128, K = 128;

  std::vector<float>                    host_A(M * K), host_B(K * N), host_C(M * N);
  std::random_device                    rd;
  std::mt19937                          gen(rd());
  std::uniform_real_distribution<float> dis(-5, 5);
  for (auto& vec : {&host_A, &host_B}) {
#if 1
    for (auto& data : *vec) {
      data = dis(gen);
    }
#else
    if (vec == &host_A) {
      for (size_t i = 0; i < vec->size(); ++i) {
        int row            = i / K;
        int col            = i % K;
        vec->operator[](i) = (row == col);
        if (row < limit && col < limit) {
          vec->operator[](i) = row * limit + col;
        }
        else {
          vec->operator[](i) = 0;
        }
      }
    }
    if (vec == &host_B) {
      for (size_t i = 0; i < vec->size(); ++i) {
        int row = i / N;
        int col = i % N;
        if (row < limit && col < limit) {
          vec->operator[](i) = row * limit + col;
        }
        else {
          vec->operator[](i) = 0;
        }
      }
    }
#endif
  }

  float *A, *B, *C;
  for (auto& pair : {std::make_pair(host_A, &A), std::make_pair(host_B, &B), std::make_pair(host_C, &C)}) {
    const std::vector<float>& host   = pair.first;
    float*&                   device = *pair.second;
    hipMalloc(&device, sizeof(float) * host.size());
    hipMemcpy(device, host.data(), sizeof(float) * host.size(), hipMemcpyDefault);
    CHECK_CUDA_ERROR();
  }

  {
    hipMemset(C, 0, M * N * sizeof(float));
    launch_fp32_naive_mm(A, B, C, M, N, K);
    hipMemcpy(host_C.data(), C, sizeof(float) * host_C.size(), hipMemcpyDefault);
    CHECK_CUDA_ERROR();
  }

  test<half>(host_A, host_B, host_C, M, N, K);
  // test<__hip_bfloat16>(host_A, host_B, host_C, M, N, K);

  CHECK_CUDA_RETURN(hipFree(A));
  CHECK_CUDA_RETURN(hipFree(B));
  CHECK_CUDA_RETURN(hipFree(C));
  return 0;
}
